#include "hip/hip_runtime.h"
#include "YoloConfigs.h"
#include "YoloLayer.h"

using namespace Yolo;

namespace nvinfer1
{
    YoloLayerPlugin::YoloLayerPlugin(int yolo_class_num, int srcSize, const int cudaThread /*= 512*/):mThreadCount(cudaThread)
    {
        // mClassCount = CLASS_NUM;
        mClassCount = yolo_class_num;
        if(srcSize==608){
            YoloKernel yolo1 = {
                19,
                19,
                {116,90,  156,198,  373,326}
            };
            YoloKernel yolo2 = {
                38,
                38,
                {30,61,  62,45,  59,119}
            };
            YoloKernel yolo3 = {
                76,
                76,
                {10,13,  16,30,  33,23}
            };

            mYoloKernel.clear();
            mYoloKernel.push_back(yolo1);
            mYoloKernel.push_back(yolo2);
            mYoloKernel.push_back(yolo3);
            mKernelCount = mYoloKernel.size();
        }
        else{
            YoloKernel yolo1 = {
                13,
                13,
                {116,90,  156,198,  373,326}
            };
            YoloKernel yolo2 = {
                26,
                26,
                {30,61,  62,45,  59,119}
            };
            YoloKernel yolo3 = {
                52,
                52,
                {10,13,  16,30,  33,23}
            };

            mYoloKernel.clear();
            mYoloKernel.push_back(yolo1);
            mYoloKernel.push_back(yolo2);
            mYoloKernel.push_back(yolo3);
            mKernelCount = mYoloKernel.size();
        }
        
    }
    
    YoloLayerPlugin::~YoloLayerPlugin()
    {
        if(mInputBuffer)
            CUDA_CHECK(hipHostFree(mInputBuffer));

        if(mOutputBuffer)
            CUDA_CHECK(hipHostFree(mOutputBuffer));
    }
    
    // create the plugin at runtime from a byte stream
    YoloLayerPlugin::YoloLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(mYoloKernel.data(),d,kernelSize);
        d += kernelSize;

        assert(d == a + length);
    }

    void YoloLayerPlugin::serialize(void* buffer)
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(d,mYoloKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t YoloLayerPlugin::getSerializationSize()
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount) + sizeof(Yolo::YoloKernel) * mYoloKernel.size();
    }

    int YoloLayerPlugin::initialize()
    { 
        int totalCount = 0;
        for(const auto& yolo : mYoloKernel)
            totalCount += (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
        CUDA_CHECK(hipHostAlloc(&mInputBuffer, totalCount * sizeof(float), hipHostMallocDefault));

        totalCount = 0;//detection count
        for(const auto& yolo : mYoloKernel)
            totalCount += yolo.width*yolo.height * CHECK_COUNT;
        CUDA_CHECK(hipHostAlloc(&mOutputBuffer, sizeof(float) + totalCount * sizeof(Detection), hipHostMallocDefault));
        return 0;
    }
    
    Dims YoloLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalCount = 0;
        for(const auto& yolo : mYoloKernel)
            totalCount += yolo.width*yolo.height * CHECK_COUNT * sizeof(Detection) / sizeof(float);

        return Dims3(totalCount + 1, 1, 1);
    }

    void YoloLayerPlugin::forwardCpu(const float*const * inputs, float* outputs, hipStream_t stream,int batchSize)
    {
        auto Logist = [=](float data){
            return 1./(1. + exp(-data));
        };

        int totalOutputCount = 0;
            int i = 0;
        int totalCount = 0;
            for(const auto& yolo : mYoloKernel)
            {
            totalOutputCount += yolo.width*yolo.height * CHECK_COUNT * sizeof(Detection) / sizeof(float);
            totalCount += (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
            ++ i;
        }

        for (int idx = 0; idx < batchSize;idx++)
        {
            i = 0;
            float* inputData = (float *)mInputBuffer;// + idx *totalCount; //if create more batch size
            for(const auto& yolo : mYoloKernel)
            {
                int size = (LOCATIONS + 1 + mClassCount) * yolo.width*yolo.height * CHECK_COUNT;
                CUDA_CHECK(hipMemcpyAsync(inputData, (float *)inputs[i] + idx * size, size * sizeof(float), hipMemcpyDeviceToHost, stream));
                inputData += size;
                ++ i;
            }

            CUDA_CHECK(hipStreamSynchronize(stream));

            inputData = (float *)mInputBuffer ;//+ idx *totalCount; //if create more batch size
            std::vector <Detection> result;
            for (const auto& yolo : mYoloKernel)
            {
                int stride = yolo.width*yolo.height;
                for (int j = 0;j < stride ;++j)
                {
                    for (int k = 0;k < CHECK_COUNT; ++k )
                    {
                        int beginIdx = (LOCATIONS + 1 + mClassCount)* stride *k + j;
                        int objIndex = beginIdx + LOCATIONS*stride;
                        
                        //check obj
                        float objProb = Logist(inputData[objIndex]);   
                        if(objProb <= IGNORE_THRESH)
                            continue;

                        //classes
                        int classId = -1;
                        float maxProb = IGNORE_THRESH;
                        for (int c = 0;c< mClassCount;++c){
                            float cProb =  Logist(inputData[beginIdx + (5 + c) * stride]) * objProb;
                            if(cProb > maxProb){
                                maxProb = cProb;
                                classId = c;
                            }
                        }
            
                        if(classId >= 0) {
                            Detection det;
                            int row = j / yolo.width;
                            int cols = j % yolo.width;
    
                            //Location
                            det.bbox[0] = (cols + Logist(inputData[beginIdx]))/ yolo.width;
                            det.bbox[1] = (row + Logist(inputData[beginIdx+stride]))/ yolo.height;
                            det.bbox[2] = exp(inputData[beginIdx+2*stride]) * yolo.anchors[2*k];
                            det.bbox[3] = exp(inputData[beginIdx+3*stride]) * yolo.anchors[2*k + 1];
                            det.classId = classId;
                            det.prob = maxProb;

                            result.emplace_back(det);
                        }
                    }
                }

                inputData += (LOCATIONS + 1 + mClassCount) * stride * CHECK_COUNT;
            }

            
            int detCount =result.size();
            auto data = (float *)mOutputBuffer;// + idx*(totalOutputCount + 1); //if create more batch size
            float * begin = data;
            //copy count;
            data[0] = (float)detCount;
            data++;
            //copy result
            memcpy(data,result.data(),result.size()*sizeof(Detection));

            //(count + det result)
            CUDA_CHECK(hipMemcpyAsync(outputs, begin,sizeof(float) + result.size()*sizeof(Detection), hipMemcpyHostToDevice, stream));

            outputs += totalOutputCount + 1;
        }
    };

    __device__ float Logist(float data){ return 1./(1. + exp(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int stride = yoloWidth*yoloHeight;
        int bnIdx =  idx / stride;

        int curIdx = idx - stride*bnIdx;

        const float* curInput = input + bnIdx* ((LOCATIONS + 1 + classes) * stride * CHECK_COUNT);

        for (int k = 0;k < CHECK_COUNT; ++k )
        {
            int beginIdx = (LOCATIONS + 1 + classes)* stride *k + curIdx;
            int objIndex = beginIdx + LOCATIONS*stride;
            
            //check objectness
            float objProb = Logist(curInput[objIndex]);
            if(objProb <= IGNORE_THRESH)
                continue;

            int row = curIdx / yoloWidth;
            int cols = curIdx % yoloWidth;
            
            //classes
            int classId = -1;
            float maxProb = IGNORE_THRESH;
            for (int c = 0;c<classes;++c){
                float cProb =  Logist(curInput[beginIdx + (5 + c) * stride]) * objProb;
                if(cProb > maxProb){
                    maxProb = cProb;
                    classId = c;
                }
            }

            if(classId >= 0) {
                float *curOutput = output + bnIdx*outputElem;
                int resCount = (int)atomicAdd(curOutput,1);
                char* data = (char * )curOutput + sizeof(float) + resCount*sizeof(Detection);
                Detection* det =  (Detection*)(data);

                //Location
                det->bbox[0] = (cols + Logist(curInput[beginIdx]))/ yoloWidth;
                det->bbox[1] = (row + Logist(curInput[beginIdx+stride]))/ yoloHeight;
                det->bbox[2] = exp(curInput[beginIdx+2*stride]) * anchors[2*k];
                det->bbox[3] = exp(curInput[beginIdx+3*stride]) * anchors[2*k + 1];
                det->classId = classId;
                det->prob = maxProb;
            }
        }
    }
   
    void YoloLayerPlugin::forwardGpu(const float *const * inputs,float * output,hipStream_t stream,int batchSize) {
        void* devAnchor;
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        CUDA_CHECK(hipMalloc(&devAnchor,AnchorLen));

        int outputElem = 1;
        for (unsigned int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            outputElem += yolo.width*yolo.height * CHECK_COUNT * sizeof(Detection) / sizeof(float);
        }

        for(int idx = 0 ;idx < batchSize;++idx)
            CUDA_CHECK(hipMemset(output + idx*outputElem, 0, sizeof(float)));

        int numElem = 0;
        for (unsigned int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize;
	        CUDA_CHECK(hipMemcpy(devAnchor,yolo.anchors,AnchorLen,hipMemcpyHostToDevice));
            CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                    (inputs[i],output, numElem, yolo.width, yolo.height, (float *)devAnchor, mClassCount ,outputElem);
        }

        CUDA_CHECK(hipFree(devAnchor));
    }


    int YoloLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs,(float *)outputs[0],stream,batchSize);

        //CPU
        //forwardCpu((const float *const *)inputs,(float *)outputs[0],stream,batchSize);
        return 0;
    };

}
